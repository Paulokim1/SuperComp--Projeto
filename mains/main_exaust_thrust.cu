#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <algorithm>
#include <cmath>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/functional.h>
#include <thrust/copy.h>
#include <thrust/sequence.h>
#include <thrust/transform.h>
#include <thrust/extrema.h>

using std::cin;
using std::cout;
using std::endl;

struct Movie
{
    int start;
    int end;
    int category;
};

struct getMovieCount
{
    int numMovies;
    int numCategories;
    Movie *movies;
    int *maxQnt_per_category;

    __device__ __host__
    getMovieCount(int numMovies, int numCategories, Movie *movies, int *maxQnt_per_category)
        : numMovies(numMovies), numCategories(numCategories), movies(movies), maxQnt_per_category(maxQnt_per_category)
    {
    }

    __device__ __host__ int operator()(const int &x)
    {
        int schedule[24] = {0};
        int currentQnt_per_category[15] = {0};

        int count = 0;

        for (int i = 0; i < numMovies; i++)
        {
            if (x & (1 << i))
            {
                Movie &movie = movies[i];
                if (currentQnt_per_category[movie.category - 1] >= maxQnt_per_category[movie.category - 1] || movie.start > movie.end)
                    return 0;
                for (int j = movie.start; j < movie.end; j++)
                {
                    if (schedule[j] == 1)
                        return 0;
                    schedule[j] = 1;
                }
                currentQnt_per_category[movie.category - 1]++;
                count++;
            }
        }
        return count;
    }
};

int main()
{
    int n, m;
    cin >> n >> m;

    thrust::host_vector<int> maxQnt_per_category_h(m);
    thrust::host_vector<Movie> movies_h(n);

    for (int i = 0; i < m; i++)
    {
        cin >> maxQnt_per_category_h[i];
    }

    for (int i = 0; i < n; i++)
    {
        Movie &movie = movies_h[i];
        cin >> movie.start >> movie.end >> movie.category;
    }

    thrust::device_vector<int> maxQnt_per_category_d(maxQnt_per_category_h);
    thrust::device_vector<Movie> movies_d(movies_h);

    thrust::device_vector<int> possibility_d(pow(2, n));
    thrust::sequence(possibility_d.begin(), possibility_d.end());

    thrust::transform(possibility_d.begin(), possibility_d.end(), possibility_d.begin(),
                      getMovieCount(n,
                                    m,
                                    thrust::raw_pointer_cast(movies_d.data()),
                                    thrust::raw_pointer_cast(maxQnt_per_category_d.data())));

    int maxMovies = *thrust::max_element(possibility_d.begin(), possibility_d.end());

    // Print the result
    cout << "Number of movies allocated: " << maxMovies << endl;

    return 0;
}